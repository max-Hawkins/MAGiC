#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include "magic.h"
}
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/cuda_device_runtime_api.h>

#define MAX_THREADS_PER_BLOCK (1024) // For my personal desktop (2070 Super) - TODO: Change to MeerKAT size

__global__ void power_spectrum(int8_t *complex_block, int *power_block, unsigned long blocsize){
    unsigned long i = (blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x)  * 4;
    if(i > 10050){ // changes to blocsize
        return;
    }
    // TODO: use dp4a 8 bit math acceleration
    unsigned long power = complex_block[i] * complex_block[i]
                            + complex_block[i+1] * complex_block[i+1]
                            + complex_block[i+2] * complex_block[i+2]
                            + complex_block[i+3] * complex_block[i+3];

    // printf("In Kernel!\tIndex: %ld  (%d, %d), (%d, %d)  Pow: %ld \n\n", 
    //             i, complex_block[i], complex_block[i+1], complex_block[i+2], complex_block[i+3], power);
    if(i == TEST_INDEX){
        printf("In Kernel!\tIndex: %ld  (%d, %d), (%d, %d)  Pow: %ld \n\n", 
                i, complex_block[i], complex_block[i+1], complex_block[i+2], complex_block[i+3], power);
    } 
    power_block[i / 4] = power;
}

extern "C" void create_power_spectrum(int8_t *file_mmap, raw_file_t *raw_file, int num_streams){

    unsigned long grid_dim_x = raw_file->blocsize / (MAX_THREADS_PER_BLOCK);
    dim3 griddim(grid_dim_x, 1, 1);
    dim3 blockdim(MAX_THREADS_PER_BLOCK / raw_file->obsnchan, raw_file->obsnchan);
    
    hipStream_t streams[num_streams];
    int8_t *h_complex_blocks[num_streams];
    int8_t *d_complex_blocks[num_streams];
    int *d_spectra[num_streams];
    int *h_spectra[num_streams];

    // Create streams and malloc data for initial streams
    for (int i = 0; i < num_streams; ++i) {
        hipStreamCreate(&streams[i]);
            printf("Stream creation:\t%s\n", hipGetErrorString(hipGetLastError()));
    }
        
    
    for(int block = 0; block < raw_file->nblocks + 4; block+= num_streams){
        // Malloc and allocate memory for all streams
        for (int i = 0; i < num_streams; ++i) {
            unsigned long block_index = raw_file->hdr_size + (block + i) * (raw_file->hdr_size + raw_file->blocsize);
            h_complex_blocks[i] = &file_mmap[block_index];

            printf("h complex: %p\t val: %d\n",&h_complex_blocks[i], h_complex_blocks[i][1000]);

            hipMalloc(&d_complex_blocks[i], raw_file->blocsize);
                printf("CudaMalloc:\t%s\n", hipGetErrorString(hipGetLastError()));
            hipMalloc(&d_spectra[i], sizeof(int) * raw_file->blocsize / 4);
                printf("CudaMalloc:\t%s\n", hipGetErrorString(hipGetLastError()));

            hipHostAlloc(&h_complex_blocks[i], raw_file->blocsize, hipHostMallocMapped);
                printf("CudaHostAlloc:\t%s\n", hipGetErrorString(hipGetLastError()));
            hipHostAlloc(&h_spectra[i], sizeof(int) * raw_file->blocsize / 4, hipHostMallocDefault);
                printf("CudaHostAlloc:\t%s\n", hipGetErrorString(hipGetLastError()));

            hipHostGetDevicePointer((void **)&h_complex_blocks[i], (void *)h_complex_blocks[i], 0);
                printf("CudaHostAlloc:\t%s\n", hipGetErrorString(hipGetLastError()));
            printf("h complex: %p\t val: %d\n",&h_complex_blocks[i], h_complex_blocks[i][1000]);

        }

        hipDeviceSynchronize();
        // Launch streams with 1 kernel launch per stream
        for(int cur_stream = 0; cur_stream < num_streams; cur_stream++){
            int cur_block = block + cur_stream;
            if(cur_block >= raw_file->nblocks){
                printf("Block number exceeded (%d). Skipping.", cur_block);
                break;
            }

            printf("\n\n--------- Block %d  Stream %d ----------\n", block, cur_stream);

            for(int i = 500; i< 508; i += 4){
                printf("I: %i  Address: %p\n", i, &h_complex_blocks[cur_stream][i]);
                printf("(%d, %d), (%d, %d)\n\n", h_complex_blocks[cur_stream][i], h_complex_blocks[cur_stream][i+1], h_complex_blocks[cur_stream][i+2], h_complex_blocks[cur_stream][i+3]);
            }
            hipMemcpyAsync(d_complex_blocks[cur_stream], h_complex_blocks[cur_stream], raw_file->blocsize, hipMemcpyHostToDevice, streams[cur_stream]);
                printf("CudaMemcpy:\t%s\n", hipGetErrorString(hipGetLastError()));

            power_spectrum<<<griddim, blockdim, 0, streams[cur_stream]>>>(d_complex_blocks[cur_stream], d_spectra[cur_stream], raw_file->blocsize);
                printf("Kernel launch:\t%s\n", hipGetErrorString(hipGetLastError()));
            hipMemcpyAsync(h_spectra[cur_stream], d_spectra[cur_stream], sizeof(int) * raw_file->blocsize / 4, hipMemcpyDeviceToHost, streams[cur_stream]);
                printf("CudaMemcpy:\t%s\n", hipGetErrorString(hipGetLastError()));
            // printf("Block: %d  Index: %ld  Contents: %d\n", block, block_index, block_address);
            // printf("Block: %d  Index: %d  Contents: %d\n", block, TEST_INDEX, file_mmap[block_index + TEST_INDEX]);
        }
        hipDeviceSynchronize();

        // Write data - TODO: implement with callbacks
        for (int i = 0; i < num_streams; ++i) {
            int cur_block = block + i;
            char *save_block_append = (char *) malloc(50);
            if(sprintf(save_block_append, "_block%03d_power.dat", cur_block) < 0){
                printf("Error creating save_filename. Couldn't save file.");
            }
            else {
                char *save_filename = (char *) malloc(70);
                strcpy(save_filename, raw_file->trimmed_filename);
                strcat(save_filename, save_block_append);

                FILE *f = fopen(save_filename, "wb");
                int status = fwrite(h_spectra[i], sizeof(int), raw_file->blocsize / 4, f);
                if(!status){
                    perror("Error writing array to file!");
                }
                fclose(f);
                free(save_filename);
            }
            free(save_block_append);

            hipFree(d_complex_blocks[i]);
            hipFree(d_spectra[i]);
            hipHostFree(h_spectra[i]);
            hipHostFree(h_complex_blocks[i]);
        }
    

    }

    // for(int i = 0; i< 100 / 4; ++i){
    //     if(h_complex_block[i]){
    //         printf("data %d: %d\n", i, h_complex_block[i]);
    //     }
    // }

    
       // printf("CudaHostAlloc:\t%s\n", hipGetErrorString(hipGetLastError()));
    
        // printf("CudaMemcpy:\t%s\n", hipGetErrorString(hipGetLastError()));
    
    // for(int i = 0; i< raw_file->blocsize / 4; ++i){
    //     if(h_spectrum[i]){
    //         printf("H_Spectrum %d: %d\n", i, h_spectrum[i]);
    //     }
    // }
    // printf("After Kernel!\tH_Complex (%d, %d), (%d, %d)\n", 
    //                 h_complex_block[TEST_INDEX], h_complex_block[TEST_INDEX+1], h_complex_block[TEST_INDEX+2], h_complex_block[TEST_INDEX+3]);
    // printf("After Kernel!\tH_Spectrum %d: %d\n", TEST_INDEX / 4, h_spectrum[TEST_INDEX / 4]);

    // Save individual block arrays to file
    
    
    for (int i = 0; i < num_streams; ++i)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    
        // printf("CudaFree:\t%s\n", hipGetErrorString(hipGetLastError()));

    
}

extern "C" void get_device_info(){
    int devCount;
    int current_device;
    hipGetDeviceCount(&devCount);
    hipGetDevice(&current_device);

    printf("-------------- CUDA Device Query ---------------\n");
    printf("CUDA devices: %d\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
                if(i == current_device){
            printf("--- Device being used ---\n");
        }
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("Device %d: %s\n", i, devProp.name);
    }
    hipSetDeviceFlags(hipDeviceMapHost);
	hipFree(0);
    printf("------------------------------------------------\n\n");
}
